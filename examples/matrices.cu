#include "../cudatrix.cu"
#include "stdio.h"
#include <vector>
#include <algorithm>

int main()
{
    // initialize matrices
    float **matrix1;
    float **matrix2;
    float **resultantMatrix;
    int rows = 2;
    int columns = 2;
    int rows1 = 2;
    int columns1 = 2;
    matrix1 = new float *[rows];
    matrix2 = new float *[rows];
    resultantMatrix = new float *[rows];

    for (int i = 0; i < rows; i++)
    {
        matrix1[i] = new float[columns];
        matrix2[i] = new float[columns];
        resultantMatrix[i] = new float[columns];
    }
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            matrix1[i][j] = 100.0f; // initialize matrix1 with values of 2
            matrix2[i][j] = 20.0f;  // initialize matrix2 with values of 1000
        }
    }
    cudatrix::matrixSum(matrix1, matrix2, resultantMatrix, rows, rows1, columns, columns1);
    printf("Matrix 1: \n");
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            printf("%f ", matrix1[i][j]);
        }
        printf("\n");
    }
    printf("Matrix 2: \n");
    for (int i = 0; i < rows1; i++)
    {
        for (int j = 0; j < columns1; j++)
        {
            printf("%f ", matrix2[i][j]); // successful sum!
        }
        printf("\n");
    }
    printf("-----------------------------\n");
    printf("\nSum of both matrices:\n");
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            printf("%f ", resultantMatrix[i][j]); // successful sum!
        }
        printf("\n");
    }
}