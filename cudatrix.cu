#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <hip/hip_runtime.h>


__global__ void scalarSumKernel(float *scalar1, float *scalar2)
{
  scalar1[threadIdx.x] += scalar2[threadIdx.x];
}

__global__ void scalarMultKernel(float *scalar1, float *scalar2)
{
  scalar1[threadIdx.x] *= scalar2[threadIdx.x];
}

void cudatrix::scalarSum(float *a, float *b)
{
  float *a_d;
  float *b_d;
  hipMallocManaged(&a_d, 1 * sizeof(float));
  hipMemcpy(a_d, a, 1 * sizeof(float), hipMemcpyHostToDevice);
  hipMallocManaged(&b_d, 1 * sizeof(float));
  hipMemcpy(b_d, b, 1 * sizeof(float), hipMemcpyHostToDevice);

  scalarSumKernel<<<1, 1>>>(a_d, b_d);
  hipDeviceSynchronize();
  hipMemcpy(a, a_d, 1 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(b, b_d, 1 * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(a_d);
  hipFree(b_d);
}

void cudatrix::scalarMult(float *a, float *b)
{
  float *a_d;
  float *b_d;
  hipMallocManaged(&a_d, 1 * sizeof(float));
  hipMemcpy(a_d, a, 1 * sizeof(float), hipMemcpyHostToDevice);
  hipMallocManaged(&b_d, 1 * sizeof(float));
  hipMemcpy(b_d, b, 1 * sizeof(float), hipMemcpyHostToDevice);
  scalarMultKernel<<<1, 1>>>(a_d, b_d);
  hipDeviceSynchronize();
  hipMemcpy(a, a_d, 1 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(b, b_d, 1 * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(a_d);
  hipFree(b_d);
}

void cudatrix::matrixSum(float **matrix1, float **matrix2, float **resultantMatrix, int rows_1, int rows_2, int columns_1, int columns_2)
{
  cudatrix::Scalar value1;
  cudatrix::Scalar value2;

  for (int i = 0; i < rows_1; i++)
  {
    for (int j = 0; j < columns_2; j++)
    {
      resultantMatrix[i][j] = 0.0;
      for (int k = 0; k < rows_2; k++)
      {
        value1.value = matrix1[i][k];
        value2.value = matrix2[k][j];
        cudatrix::scalarSum(&value1.value, &value2.value);
        resultantMatrix[i][j] = value1.value;
      }
    }
  }
}

void cudatrix::matrixMult(float **matrix1, float **matrix2, float **resultantMatrix, int rows_1, int rows_2, int columns_1, int columns_2)
{
  cudatrix::Scalar value1;
  cudatrix::Scalar value2;

  for (int i = 0; i < rows_1; i++)
  {
    for (int j = 0; j < columns_2; j++)
    {
      resultantMatrix[i][j] = 0.0;
      for (int k = 0; k < rows_2; k++)
      {
        value1.value = matrix1[i][k];
        value2.value = matrix2[k][j];
        cudatrix::scalarMult(&value1.value, &value2.value);
        resultantMatrix[i][j] = value1.value;
      }
    }
  }
}